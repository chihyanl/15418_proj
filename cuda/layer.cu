#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include "layer.h"

#define SEED 0
#define BLOCK_SIZE 256

int nextPow2(int n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  n++;
  return n;
}

__device__ __inline__ void cudaMax(float *temp, int N, int idx) {
  for (int twod = N / 2; twod >= 1; twod /= 2) {
    if (idx < twod && temp[idx + twod] > temp[idx]) {
      temp[idx] = temp[idx + twod];
    }
    __syncthreads();
  }
}

__device__ __inline__ void cudaSum(float *temp, int N, int idx) {
  for (int twod = N / 2; twod >= 1; twod /= 2) {
    if (idx < twod) {
      temp[idx] += temp[idx + twod];
    }
    __syncthreads();
  }
}

__global__ void cudaGetCorrect(int label, float *output, float *error, int N,
                               int size, int *correct) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= N) {
    return;
  }

  extern __shared__ float temp[];
  __shared__ int pred;

  temp[idx] = 0.0f;
  if (idx < size) {
    temp[idx] = output[idx];
    error[idx] = 0.0f;
  }

  if (idx == label) {
    error[idx] -= 1.0f;
  }
  __syncthreads();

  cudaMax(temp, N, idx); // reduce max

  if (idx < size && temp[0] == output[idx]) {
    pred = idx;
  }
  __syncthreads();
  if (idx == 0) {
    *correct += pred == label;
  }
}

void getCorrect(int label, float *output, float *error, int size,
                int *correct) {
  int N = nextPow2(size);

  dim3 blockDim(N, 1);
  dim3 gridDim(1);
  cudaGetCorrect<<<gridDim, blockDim, sizeof(float) * N>>>(label, output, error,
                                                           N, size, correct);
}

__global__ void randomFloat(float *ptr, float a, float b, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= size) {
    return;
  }

  hiprandState state;
  hiprand_init(SEED, idx, 0, &state);
  ptr[idx] = hiprand_uniform(&state) * (b - a) + a;
}

__global__ void cudaConvForward(int height, int width, int in_channels,
                                int out_channels, int kernel_h, int kernel_w,
                                int stride, int pad, int height_out,
                                int width_out, float *input, float *output,
                                float *weight, float *bias) {

  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out_channels * height_out * width_out) {
    return;
  }

  int dst_channel = outIdx / (height_out * width_out);
  int dst_y = (outIdx / width_out) % height_out;
  int dst_x = outIdx % width_out;

  int dstIdx_base = dst_channel * in_channels * kernel_h * kernel_w;
  int src_y = stride * dst_y - pad;
  int src_x = stride * dst_x - pad;

  float sum = bias[dst_channel];
  for (int src_channel = 0; src_channel < in_channels; src_channel++) {
    int dstIdx_base2 = src_channel * kernel_h * kernel_w;
    int srcIdx_base = src_channel * width * height;
    for (int dy = 0; dy < kernel_h; dy++) {
      int y = src_y + dy;
      if (y >= 0 && y < height) {
        int srcIdx = srcIdx_base + y * width;
        int dstIdx = dstIdx_base + dstIdx_base2 + dy * kernel_w;
        for (int dx = 0; dx < kernel_w; dx++) {
          int x = src_x + dx;
          if (x >= 0 && x < width) {
            sum += input[srcIdx + x] * weight[dstIdx + dx];
          }
        }
      }
    }
  }

  sum = (sum > 0) ? sum : 0; // relu
  output[outIdx] = sum;
}

__global__ void cudaConvBackward(int height, int width, int in_channels,
                                 int out_channels, int kernel_h, int kernel_w,
                                 int stride, int pad, int height_out,
                                 int width_out, float *input, float *output,
                                 float *weight, float *u_weight, float *u_bias,
                                 float *error, float *src_error) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out_channels * height_out * width_out) {
    return;
  }

  int dst_channel = outIdx / (height_out * width_out);
  int dst_y = (outIdx / width_out) % height_out;
  int dst_x = outIdx % width_out;

  int dstIdx_base = dst_channel * in_channels * kernel_h * kernel_w;
  int src_y = stride * dst_y - pad;
  int src_x = stride * dst_x - pad;

  float dnet = error[outIdx] * (output[outIdx] > 0);
  atomicAdd(&u_bias[dst_channel], dnet);
  for (int src_channel = 0; src_channel < in_channels; src_channel++) {
    int dstIdx_base2 = src_channel * kernel_h * kernel_w;
    int srcIdx_base = src_channel * width * height;
    for (int dy = 0; dy < kernel_h; dy++) {
      int y = src_y + dy;
      if (y >= 0 && y < height) {
        int srcIdx = srcIdx_base + y * width;
        int dstIdx = dstIdx_base + dstIdx_base2 + dy * kernel_h;
        for (int dx = 0; dx < kernel_w; dx++) {
          int x = src_x + dx;
          if (x >= 0 && x < width) {
            if (src_error != nullptr) {
              atomicAdd(&src_error[srcIdx + x], weight[dstIdx + dx] * dnet);
            }
            atomicAdd(&u_weight[dstIdx + dx], dnet * input[srcIdx + x]);
          }
        }
      }
    }
  }
}

__global__ void cudaConvUpdate(int in_channels, int out_channels, int kernel_h,
                               int kernel_w, float *weight, float *u_weight,
                               float *bias, float *u_bias, float rate) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels * in_channels * kernel_h * kernel_w) {
    return;
  }

  weight[i] -= rate * u_weight[i];
  u_weight[i] = 0;

  if (i >= out_channels) {
    return;
  }

  bias[i] -= rate * u_bias[i];
  u_bias[i] = 0;
}

__global__ void cudaConvFusionForward(
    int in_channels, int mid_channels, int out_channels, int in_height,
    int in_width, int l1_kernel_h, int l1_kernel_w, int l1_stride, int l1_pad,
    int l1_h_out, int l1_w_out, int l2_kernel_h, int l2_kernel_w, int l2_stride,
    int l2_pad, int l2_h_out, int l2_w_out, float *input, float *l1_output,
    float *l2_output, float *l1_weight, float *l2_weight, float *l1_bias,
    float *l2_bias) {
  // TODO: implment this lol this is a lotta params
  return;
}

__global__ void cudaConvFusionBackward(
    int in_height, int in_width, int in_channels, int mid_channels,
    int out_channels, int l1_kernel_h, int l1_kernel_w, int l1_stride,
    int l1_pad, int l1_h_out, int l1_w_out, int l2_kernel_h, int l2_kernel_w,
    int l2_stride, int l2_pad, int l2_h_out, int l2_w_out, float *input,
    float *l1_output, float *l1_weight, float *l1_u_weight, float *l1_u_bias,
    float *l1_error, float *l2_output, float *l2_weight, float *l2_u_weight,
    float *l2_u_bias, float *l2_error, float *src_error) {
  // TODO: implement this
  return;
}

__global__ void cudaConvFusionUpdate(
    int in_channels, int mid_channels, int out_channels, int l1_kernel_h,
    int l1_kernel_w, int l2_kernel_h, int l2_kernel_w, float *l1_weight,
    float *l1_u_weight, float *l1_bias, float *l1_u_bias, float *l2_weight,
    float *l2_u_weight, float *l2_bias, float *l2_u_bias, float rate) {
  // TODO: implement this
  return;
}

__global__ void cudaLinearForward(int in_channels, int out_channels,
                                  float *input, float *weight, float *bias,
                                  float *output, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N) {
    return;
  }

  extern __shared__ float temp[];

  float x = 0.0f;
  if (i < out_channels) {
    x = bias[i];
    for (int j = 0; j < in_channels; j++) {
      x += input[j] * weight[i * in_channels + j];
    }
    output[i] = x;
  }
  temp[i] = x;
  __syncthreads();

  // softmax
  cudaMax(temp, N, i); // reduce max

  x = 0.0f;
  if (i < out_channels) {
    x = expf(output[i] - temp[0]);
    output[i] = x;
  }
  temp[i] = x;
  __syncthreads();

  cudaSum(temp, N, i); // reduce sum
  float sum = temp[0] + 0.0001;

  // softmax
  if (i < out_channels) {
    output[i] /= sum;
  }
}

__global__ void cudaLinearBackward(int in_channels, int out_channels,
                                   float *input, float *weight, float *u_weight,
                                   float *u_bias, float *output, float *error,
                                   float *src_error) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels) {
    return;
  }

  float gradient = output[i] + error[i];
  for (int j = 0; j < in_channels; j++) {
    atomicAdd(&src_error[j], weight[i * in_channels + j] * gradient);
    u_weight[i * in_channels + j] += gradient * input[j];
  }
  u_bias[i] += gradient;
}

__global__ void cudaLinearUpdate(int in_channels, int out_channels,
                                 float *weight, float *u_weight, float *bias,
                                 float *u_bias, float *error, float rate) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels * in_channels) {
    return;
  }

  weight[i] -= rate * u_weight[i];
  u_weight[i] = 0.0f;

  if (i >= out_channels) {
    return;
  }

  bias[i] -= rate * u_bias[i];
  u_bias[i] = 0.0f;
}

Conv::Conv(int in_channels, int out_channels, int height, int width,
           int kernel_h, int kernel_w, int stride, int pad)
    : height(height),             // input height
      width(width),               // input width
      in_channels(in_channels),   // number of input channels
      out_channels(out_channels), // number of output channels
      kernel_h(kernel_h),         // filter height
      kernel_w(kernel_w),         // filter width
      stride(stride), pad(pad),
      height_out((height + 2 * pad - kernel_h) / stride + 1),
      width_out((width + 2 * pad - kernel_w) / stride + 1) {

  hipMalloc(&output, sizeof(float) * out_channels * height_out * width_out);
  hipMalloc(&bias, sizeof(float) * out_channels);
  hipMalloc(&weight,
             sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);

  hipMemset(output, 0, sizeof(float) * out_channels * height_out * width_out);
  hipMemset(bias, 0, sizeof(float) * out_channels);
  hipMemset(weight, 0,
             sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);

  hipMalloc(&u_bias, sizeof(float) * out_channels);
  hipMalloc(&u_weight,
             sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);
  hipMalloc(&error, sizeof(float) * out_channels * height_out * width_out);

  hipMemset(u_bias, 0, sizeof(float) * out_channels);
  hipMemset(u_weight, 0,
             sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);
  hipMemset(error, 0, sizeof(float) * out_channels * height_out * width_out);

  printf("Conv=%dx%dx%d (%d->%d), Kernel=%dx%d, Stride=%d, Pad=%d, Weight=%d, "
         "Bias=%d\n",
         height_out, width_out, out_channels, in_channels * width * height,
         out_channels * height_out * width_out, kernel_h, kernel_w, stride, pad,
         out_channels * in_channels * kernel_h * kernel_w, out_channels);

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim(
      (out_channels * in_channels * kernel_h * kernel_w + blockDim.x - 1) /
      blockDim.x);

  float k = sqrt(1.0f / (in_channels * kernel_h * kernel_w));
  randomFloat<<<gridDim, blockDim>>>(
      weight, -k, k, out_channels * in_channels * kernel_h * kernel_w);
  randomFloat<<<gridDim, blockDim>>>(bias, -k, k, out_channels);
}

Conv::~Conv() {
  hipFree(output);
  hipFree(bias);
  hipFree(weight);
  hipFree(error);
  hipFree(u_bias);
  hipFree(u_weight);
}

void Conv::forward(float *) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * height_out * width_out + blockDim.x - 1) /
               blockDim.x);

  cudaConvForward<<<gridDim, blockDim>>>(
      height, width, in_channels, out_channels, kernel_h, kernel_w, stride, pad,
      height_out, width_out, input, output, weight, bias);
}

void Conv::backward(float *input, float *src_error) {
  if (src_error != nullptr) {
    hipMemset(src_error, 0, sizeof(float) * in_channels * width * height);
  }

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * height_out * width_out + blockDim.x - 1) /
               blockDim.x);

  cudaConvBackward<<<gridDim, blockDim>>>(
      height, width, in_channels, out_channels, kernel_h, kernel_w, stride, pad,
      height_out, width_out, input, output, weight, u_weight, u_bias, error,
      src_error);
}

void Conv::update(float rate) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim(
      (out_channels * in_channels * kernel_h * kernel_w + blockDim.x - 1) /
      blockDim.x);

  cudaConvUpdate<<<gridDim, blockDim>>>(in_channels, out_channels, kernel_h,
                                        kernel_w, weight, u_weight, bias,
                                        u_bias, rate);
}

ConvFuse::ConvFuse(int in_channels, int mid_channels, int out_channels,
                   ConvLayerConfig &l1_config, ConvLayerConfig &l2_config)
    : in_height(l1_config.height), in_width(l1_config.width),
      in_channels(in_channels), mid_channels(mid_channels),
      out_channels(out_channels), l1_config(l1_config), l2_config(l2_config),
      l1_h_out((l1_config.height + 2 * l1_config.pad - l1_config.kernel_h) /
                   l1_config.stride +
               1),
      l1_w_out((l1_config.width + 2 * l1_config.pad - l1_config.kernel_w) /
                   l1_config.stride +
               1),
      l2_h_out((l2_config.height + 2 * l2_config.pad - l2_config.kernel_h) /
                   l2_config.stride +
               1),
      l2_w_out((l2_config.width + 2 * l2_config.pad - l2_config.kernel_w) /
                   l2_config.stride +
               1) {

  // All memory needed for first layer
  hipMalloc(&mid_layer.output,
             sizeof(float) * mid_channels * l1_h_out * l1_w_out);
  hipMalloc(&mid_layer.bias, sizeof(float) * mid_channels);
  hipMalloc(&mid_layer.weight, sizeof(float) * mid_channels * in_channels *
                                    l1_config.kernel_h * l1_config.kernel_w);

  hipMemset(mid_layer.output, 0,
             sizeof(float) * mid_channels * l1_h_out * l1_w_out);
  hipMemset(mid_layer.bias, 0, sizeof(float) * mid_channels);
  hipMemset(mid_layer.weight, 0,
             sizeof(float) * mid_channels * in_channels * l1_config.kernel_h *
                 l1_config.kernel_w);

  hipMalloc(&mid_layer.u_bias, sizeof(float) * mid_channels);
  hipMalloc(&mid_layer.u_weight, sizeof(float) * mid_channels * in_channels *
                                      l1_config.kernel_h * l1_config.kernel_w);
  hipMalloc(&mid_layer.error,
             sizeof(float) * mid_channels * l1_h_out * l1_w_out);

  hipMemset(mid_layer.u_bias, 0, sizeof(float) * mid_channels);
  hipMemset(mid_layer.u_weight, 0,
             sizeof(float) * mid_channels * in_channels * l1_config.kernel_h *
                 l1_config.kernel_w);
  hipMemset(mid_layer.error, 0,
             sizeof(float) * mid_channels * l1_h_out * l1_w_out);

  printf("ConvFusionL1=%dx%dx%d (%d->%d), Kernel=%dx%d, Stride=%d, Pad=%d, "
         "Weight=%d, Bias=%d\n",
         l1_h_out, l1_w_out, mid_channels,
         in_channels * l1_config.width * l1_config.height,
         mid_channels * l1_config.height_out * l1_config.width_out,
         l1_config.kernel_h, l1_config.kernel_w, l1_config.stride,
         l1_config.pad,
         mid_channels * in_channels * l1_config.kernel_h * l1_config.kernel_w,
         mid_channels);

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim(
      (mid_channels * in_channels * l1_config.kernel_h * l1_config.kernel_w +
       blockDim.x - 1) /
      blockDim.x);
  float k =
      sqrt(1.0f / (in_channels * l1_config.kernel_h * l1_config.kernel_w));
  randomFloat<<<gridDim, blockDim>>>(
      mid_layer.weight, -k, k,
      mid_channels * in_channels * l1_config.kernel_h * l1_config.kernel_w);
  randomFloat<<<gridDim, blockDim>>>(mid_layer.bias, -k, k, mid_channels);

  // All memory needed for final layer
  hipMalloc(&output, sizeof(float) * out_channels * l2_h_out * l2_w_out);
  hipMalloc(&bias, sizeof(float) * out_channels);
  hipMalloc(&weight, sizeof(float) * out_channels * mid_channels *
                          l2_config.kernel_h * l2_config.kernel_w);

  hipMemset(output, 0, sizeof(float) * out_channels * l2_h_out * l2_w_out);
  hipMemset(bias, 0, sizeof(float) * out_channels);
  hipMemset(weight, 0,
             sizeof(float) * out_channels * mid_channels * l2_config.kernel_h *
                 l2_config.kernel_w);

  hipMalloc(&u_bias, sizeof(float) * out_channels);
  hipMalloc(&u_weight, sizeof(float) * out_channels * mid_channels *
                            l2_config.kernel_h * l2_config.kernel_w);
  hipMalloc(&error, sizeof(float) * out_channels * l2_h_out * l2_w_out);

  hipMemset(u_bias, 0, sizeof(float) * out_channels);
  hipMemset(u_weight, 0,
             sizeof(float) * out_channels * mid_channels * l2_config.kernel_h *
                 l2_config.kernel_w);
  hipMemset(error, 0, sizeof(float) * out_channels * l2_h_out * l2_w_out);

  printf("ConvFusionL2=%dx%dx%d (%d->%d), Kernel=%dx%d, Stride=%d, Pad=%d, "
         "Weight=%d, Bias=%d\n",
         l2_h_out, l2_w_out, out_channels,
         mid_channels * l2_config.width * l2_config.height,
         out_channels * l2_config.height_out * l2_config.width_out,
         l2_config.kernel_h, l2_config.kernel_w, l2_config.stride,
         l2_config.pad,
         out_channels * mid_channels * l2_config.kernel_h * l2_config.kernel_w,
         out_channels);

  dim3 gridDim2(
      (out_channels * mid_channels * l2_config.kernel_h * l2_config.kernel_w +
       blockDim.x - 1) /
      blockDim.x);
  float k =
      sqrt(1.0f / (mid_channels * l2_config.kernel_h * l2_config.kernel_w));
  randomFloat<<<gridDim2, blockDim>>>(
      weight, -k, k,
      out_channels * mid_channels * l2_config.kernel_h * l2_config.kernel_w);
  randomFloat<<<gridDim2, blockDim>>>(bias, -k, k, out_channels);
}

ConvFuse::~ConvFuse() {
  hipFree(mid_layer.output);
  hipFree(mid_layer.bias);
  hipFree(mid_layer.weight);
  hipFree(mid_layer.error);
  hipFree(mid_layer.u_bias);
  hipFree(mid_layer.u_weight);
  hipFree(output);
  hipFree(bias);
  hipFree(weight);
  hipFree(error);
  hipFree(u_bias);
  hipFree(u_weight);
}

void ConvFuse::forward(float *input) {
  dim3 blockDim(BLOCK_SIZE, 1);
  // TODO: what is proper griddim
  dim3 gridDim((out_channels * l2_h_out * l2_w_out + blockDim.x - 1) /
               blockDim.x);

  cudaConvFusionForward<<<gridDim, blockDim>>>(
      in_channels, mid_channels, out_channels, in_height, in_width,
      l1_config.kernel_h, l1_config.kerneh_w, l1_config.stride, l1_config.pad,
      l1_h_out, l1_w_out, l2_config.kernel_h, l2_config.kernel_w,
      l2_config.stride, l2_config.pad, l2_h_out, l2_w_out, intput,
      mid_layer.output, output, mid_layer.weight, weight, mid_layer.bias, bias);
}

void ConvFuse::backward(float *input, float *src_error) {
  if (src_error != nullptr) {
    hipMemset(src_error, 0,
               sizeof(float) * in_channels * in_width * in_height);
  }

  dim3 blockDim(BLOCK_SIZE, 1);
  // TODO: what is proper griddim
  dim3 gridDim((out_channels * l2_config.height_out * l2_config.width_out +
                blockDim.x - 1) /
               blockDim.x);

  cudaConvFusionBackward<<<gridDim, blockDim>>>(
      in_height, in_width, in_channels, mid_channels, out_channels,
      l1_config.kernel_h, l1_config.kernel_w, l1_config.stride, l1_config.pad,
      l1_h_out, l1_w_out, l2_config.kernel_h, l2_config.kernel_w,
      l2_config.stride, l2_config.pad, l2_h_out, l2_w_out, input,
      mid_layer.output, mid_layer.weight, mid_layer.u_weight, mid_layer.u_bias,
      mid_layer.error, output, weight, u_weight, u_bias, error, src_error);
}

void ConvFusion::update(float rate) {
  dim3 blockDim(BLOCK_SIZE, 1);
  // TODO: what is proper griddim
  dim3 gridDim(
      (out_channels * in_channels * kernel_h * kernel_w + blockDim.x - 1) /
      blockDim.x);

  cudaConvFusionUpdate<<<gridDim, blockDim>>>(
      in_channels, mid_channels, out_channels, l1_config.kernel_h,
      l1_config.kernel_w, l2_config.kernel_h, l2_config.kernel_w,
      mid_layer.weight, mid_layer.u_weight, mid_layer.bias, mid_layer.u_bias,
      weight, u_weight, bias, u_bias, rate);
}

Linear::Linear(int in_channels, int out_channels)
    : in_channels(in_channels), out_channels(out_channels) {

  hipMalloc(&output, sizeof(float) * out_channels);
  hipMalloc(&bias, sizeof(float) * out_channels);
  hipMalloc(&weight, sizeof(float) * in_channels * out_channels);

  hipMemset(output, 0, sizeof(float) * out_channels);
  hipMemset(bias, 0, sizeof(float) * out_channels);
  hipMemset(weight, 0, sizeof(float) * in_channels * out_channels);

  hipMalloc(&error, sizeof(float) * out_channels);
  hipMalloc(&u_bias, sizeof(float) * out_channels);
  hipMalloc(&u_weight, sizeof(float) * out_channels * in_channels);

  hipMemset(error, 0, sizeof(float) * out_channels);
  hipMemset(u_bias, 0, sizeof(float) * out_channels);
  hipMemset(u_weight, 0, sizeof(float) * out_channels * in_channels);

  printf("Linear=%d->%d\n", in_channels, out_channels);

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * in_channels + blockDim.x - 1) / blockDim.x);

  float k = 1.0f / in_channels;
  randomFloat<<<gridDim, blockDim>>>(weight, -k, k, out_channels * in_channels);
  randomFloat<<<gridDim, blockDim>>>(bias, -k, k, out_channels);
}

Linear::~Linear() {
  hipFree(output);
  hipFree(bias);
  hipFree(weight);
  hipFree(error);
  hipFree(u_bias);
  hipFree(u_weight);
}

void Linear::forward(float *input) {
  int N = nextPow2(out_channels);

  if (N > BLOCK_SIZE) {
    printf("Error: Linear Layer out_channels greater than BLOCK_SIZE");
    return;
  }

  dim3 blockDim(N, 1);
  dim3 gridDim(1);
  cudaLinearForward<<<gridDim, blockDim, sizeof(float) * N>>>(
      in_channels, out_channels, input, weight, bias, output, N);
}

void Linear::backward(float *input, float *src_error) {
  hipMemset(src_error, 0, sizeof(float) * in_channels);

  dim3 blockDim(out_channels, 1);
  dim3 gridDim(1);
  cudaLinearBackward<<<gridDim, blockDim>>>(in_channels, out_channels, input,
                                            weight, u_weight, u_bias, output,
                                            error, src_error);
}

void Linear::update(float rate) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((in_channels * out_channels + BLOCK_SIZE - 1) / BLOCK_SIZE);
  cudaLinearUpdate<<<gridDim, blockDim>>>(in_channels, out_channels, weight,
                                          u_weight, bias, u_bias, error, rate);
}
