#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdlib.h>
#include <cmath>
#include <stdio.h>

#include "layer.h"

#define SEED 0
#define BLOCK_SIZE 256

int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__device__ __inline__ void cudaMax(float* temp, int N, int idx) {
  for (int twod = N/2; twod >= 1; twod /= 2) {
    if (idx < twod && temp[idx + twod] > temp[idx]) {
      temp[idx] = temp[idx + twod];
    }
    __syncthreads();
  }
}

__device__ __inline__ void cudaSum(float* temp, int N, int idx) {
  for (int twod = N/2; twod >= 1; twod /= 2) {
    if (idx < twod) {
      temp[idx] += temp[idx + twod];
    }
    __syncthreads();
  }
}

__global__ void cudaGetCorrect(int label, float* output, float* error, int N, int size, int* correct) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= N) {
    return;
  }

  extern __shared__ float temp[];
  __shared__ int pred;

  temp[idx] = 0.0f;
  if (idx < size) {
    temp[idx] = output[idx];
    error[idx] = 0.0f;
  }
  
  if (idx == label) {
    error[idx] -= 1.0f;
  }
  __syncthreads();

  cudaMax(temp, N, idx);  // reduce max

  if (idx < size && temp[0] == output[idx]) {
    pred = idx;
  }
  __syncthreads();
  if (idx == 0) {
    *correct += pred == label;
  }
}

void getCorrect(int label, float* output, float* error, int size, int* correct) {
  int N = nextPow2(size);

  dim3 blockDim(N, 1);
  dim3 gridDim(1);
  cudaGetCorrect<<<gridDim, blockDim, sizeof(float) * N>>>(label, output, error, N, size, correct);
}

__global__ void randomFloat(float* ptr, float a, float b, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= size) {
    return;
  }

  hiprandState state;
  hiprand_init(SEED, idx, 0, &state);
  ptr[idx] = hiprand_uniform(&state) * (b - a) + a;
}

__global__ void cudaConvForward(int height, int width, int in_channels, int out_channels, int kernel_h, int kernel_w, int stride, int pad, int height_out, int width_out, float* input, float* output, float* weight, float* bias) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out_channels * height_out * width_out) {
    return;
  }

  int dst_channel = outIdx / (height_out * width_out);
  int dst_y = (outIdx / width_out) % height_out;
  int dst_x = outIdx % width_out;

  int dstIdx_base = dst_channel * in_channels * kernel_h * kernel_w;
  int src_y = stride * dst_y - pad;
  int src_x = stride * dst_x - pad;

  float sum = bias[dst_channel];
  for (int src_channel = 0; src_channel < in_channels; src_channel++) {
    int dstIdx_base2 = src_channel * kernel_h * kernel_w;
    int srcIdx_base = src_channel * width * height;
    for (int dy = 0; dy < kernel_h; dy++) {
      int y = src_y + dy;
      if (y >= 0 && y < height) {
        int srcIdx = srcIdx_base + y * width;
        int dstIdx = dstIdx_base + dstIdx_base2 + dy * kernel_w;
        for (int dx = 0; dx < kernel_w; dx++) {
          int x = src_x + dx;
          if (x >= 0 && x < width) {
            sum += input[srcIdx+x] * weight[dstIdx+dx];
          }
        }
      }
    }
  }
  
  sum = (sum > 0) ? sum : 0;  // relu
  output[outIdx] = sum;
}

__global__ void cudaConvBackward(int height, int width, int in_channels, int out_channels, int kernel_h, int kernel_w, int stride, int pad, int height_out, int width_out, float* input, float* output, float* weight, float* u_weight, float* u_bias, float* error, float* src_error) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (outIdx >= out_channels * height_out * width_out) {
    return;
  }

  int dst_channel = outIdx / (height_out * width_out);
  int dst_y = (outIdx / width_out) % height_out;
  int dst_x = outIdx % width_out;

  int dstIdx_base = dst_channel * in_channels * kernel_h * kernel_w;
  int src_y = stride * dst_y - pad;
  int src_x = stride * dst_x - pad;

  float dnet = error[outIdx] * (output[outIdx] > 0);
  atomicAdd(&u_bias[dst_channel], dnet);
  for (int src_channel = 0; src_channel < in_channels; src_channel++) {
    int dstIdx_base2 = src_channel * kernel_h * kernel_w;
    int srcIdx_base = src_channel * width * height;
    for (int dy = 0; dy < kernel_h; dy++) {
      int y = src_y + dy;
      if (y >= 0 && y < height) {
        int srcIdx = srcIdx_base + y * width;
        int dstIdx = dstIdx_base + dstIdx_base2 + dy * kernel_h;
        for (int dx = 0; dx < kernel_w; dx++) {
          int x = src_x + dx;
          if (x >= 0 && x < width) {
            if (src_error != nullptr) {
              atomicAdd(&src_error[srcIdx+x], weight[dstIdx+dx] * dnet);
            }
            atomicAdd(&u_weight[dstIdx+dx], dnet * input[srcIdx+x]);
          }
        }
      }
    }
  }
}

__global__ void cudaConvUpdate(int in_channels, int out_channels, int kernel_h, int kernel_w, float* weight, float* u_weight, float* bias, float* u_bias, float rate) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels * in_channels * kernel_h * kernel_w) {
    return;
  }

  weight[i] -= rate * u_weight[i];
  u_weight[i] = 0;

  if (i >= out_channels) {
    return;
  }

  bias[i] -= rate * u_bias[i];
  u_bias[i] = 0;
}

__global__ void cudaLinearForward(int in_channels, int out_channels, float* input, float* weight, float* bias, float* output, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N) {
    return;
  }

  extern __shared__ float temp[];

  float x = 0.0f;
  if (i < out_channels) {
    x = bias[i];
    for (int j = 0; j < in_channels; j++) {
      x += input[j] * weight[i*in_channels+j];
    }
    output[i] = x;
  }
  temp[i] = x;
  __syncthreads();

  // softmax 
  cudaMax(temp, N, i);  // reduce max

  x = 0.0f;
  if (i < out_channels) {
    x = expf(output[i] - temp[0]);
    output[i] = x;
  }
  temp[i] = x;
  __syncthreads();

  cudaSum(temp, N, i);  // reduce sum
  float sum = temp[0] + 0.0001; 
  
  //softmax
  if (i < out_channels) {
    output[i] /= sum;
  }
}

__global__ void cudaLinearBackward(int in_channels, int out_channels, float* input, float* weight, float* u_weight, float* u_bias, float* output, float* error, float* src_error) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels) {
    return;
  }

  float gradient = output[i] + error[i];
  for (int j = 0; j < in_channels; j++) {
    atomicAdd(&src_error[j], weight[i*in_channels+j] * gradient);
    u_weight[i*in_channels+j] += gradient * input[j];
  }
  u_bias[i] += gradient;
}

__global__ void cudaLinearUpdate(int in_channels, int out_channels, float* weight, float* u_weight, float* bias, float* u_bias, float* error, float rate) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= out_channels * in_channels) {
    return;
  }

  weight[i] -= rate * u_weight[i];
  u_weight[i] = 0.0f;

  if (i >= out_channels) {
    return;
  }

  bias[i] -= rate * u_bias[i];
  u_bias[i] = 0.0f;
}

Conv::Conv(int in_channels, int out_channels, int height, int width, int kernel_h, int kernel_w, int stride, int pad)
  : height(height),               // input height
    width(width),                 // input width
    in_channels(in_channels),     // number of input channels
    out_channels(out_channels),   // number of output channels
    kernel_h(kernel_h),           // filter height
    kernel_w(kernel_w),           // filter width
    stride(stride),
    pad(pad),
    height_out((height + 2 * pad - kernel_h) / stride + 1),
    width_out((width + 2 * pad - kernel_w) / stride + 1) {

  hipMalloc(&output, sizeof(float) * out_channels * height_out * width_out);
  hipMalloc(&bias, sizeof(float) * out_channels);
  hipMalloc(&weight, sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);

  hipMemset(output, 0, sizeof(float) * out_channels * height_out * width_out);
  hipMemset(bias, 0, sizeof(float) * out_channels);
  hipMemset(weight, 0, sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);

  hipMalloc(&u_bias, sizeof(float) * out_channels);
  hipMalloc(&u_weight, sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);
  hipMalloc(&error, sizeof(float) * out_channels * height_out * width_out);
  
  hipMemset(u_bias, 0, sizeof(float) * out_channels);
  hipMemset(u_weight, 0, sizeof(float) * out_channels * in_channels * kernel_h * kernel_w);
  hipMemset(error, 0, sizeof(float) * out_channels * height_out * width_out);

  printf("Conv=%dx%dx%d (%d->%d), Kernel=%dx%d, Stride=%d, Pad=%d, Weight=%d, Bias=%d\n", height_out, width_out, out_channels, in_channels*width*height, out_channels*height_out*width_out, kernel_h, kernel_w, stride, pad, out_channels*in_channels*kernel_h*kernel_w, out_channels);

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * in_channels * kernel_h * kernel_w + blockDim.x - 1) / blockDim.x);
  
  float k = sqrt(1.0f / (in_channels * kernel_h * kernel_w));
  randomFloat<<<gridDim, blockDim>>>(weight, -k, k, out_channels * in_channels * kernel_h * kernel_w);
  randomFloat<<<gridDim, blockDim>>>(bias, -k, k, out_channels);
}

Conv::~Conv() {
  hipFree(output);
  hipFree(bias);
  hipFree(weight);
  hipFree(error);
  hipFree(u_bias);
  hipFree(u_weight);
}

void Conv::forward(float* input) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * height_out * width_out + blockDim.x - 1) / blockDim.x);

  cudaConvForward<<<gridDim, blockDim>>>(height, width, in_channels, out_channels, kernel_h, kernel_w, stride, pad, height_out, width_out, input, output, weight, bias);
}

void Conv::backward(float* input, float* src_error) {
  if (src_error != nullptr) {
    hipMemset(src_error, 0, sizeof(float) * in_channels * width * height);
  }

  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * height_out * width_out + blockDim.x - 1) / blockDim.x);

  cudaConvBackward<<<gridDim, blockDim>>>(height, width, in_channels, out_channels, kernel_h, kernel_w, stride, pad, height_out, width_out, input, output, weight, u_weight, u_bias, error, src_error);
}

void Conv::update(float rate) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * in_channels * kernel_h * kernel_w + blockDim.x - 1) / blockDim.x);

  cudaConvUpdate<<<gridDim, blockDim>>>(in_channels, out_channels, kernel_h, kernel_w, weight, u_weight, bias, u_bias, rate);
}


Linear::Linear(int in_channels, int out_channels)
  : in_channels(in_channels),
    out_channels(out_channels) {

  hipMalloc(&output, sizeof(float) * out_channels);
  hipMalloc(&bias, sizeof(float) * out_channels);
  hipMalloc(&weight, sizeof(float) * in_channels * out_channels);

  hipMemset(output, 0, sizeof(float) * out_channels);
  hipMemset(bias, 0, sizeof(float) * out_channels);
  hipMemset(weight, 0, sizeof(float) * in_channels * out_channels);

  hipMalloc(&error, sizeof(float) * out_channels);
  hipMalloc(&u_bias, sizeof(float) * out_channels);
  hipMalloc(&u_weight, sizeof(float) * out_channels * in_channels);

  hipMemset(error, 0, sizeof(float) * out_channels);
  hipMemset(u_bias, 0, sizeof(float) * out_channels);
  hipMemset(u_weight, 0, sizeof(float) * out_channels * in_channels);

  printf("Linear=%d->%d\n", in_channels, out_channels);
  
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((out_channels * in_channels + blockDim.x - 1) / blockDim.x);

  float k = 1.0f / in_channels;
  randomFloat<<<gridDim, blockDim>>>(weight, -k, k, out_channels * in_channels);
  randomFloat<<<gridDim, blockDim>>>(bias, -k, k, out_channels);
}

Linear::~Linear() {
  hipFree(output);
  hipFree(bias);
  hipFree(weight);
  hipFree(error);
  hipFree(u_bias);
  hipFree(u_weight);
}

void Linear::forward(float* input) {
  int N = nextPow2(out_channels);

  if (N > BLOCK_SIZE) {
    printf("Error: Linear Layer out_channels greater than BLOCK_SIZE");
    return;
  }

  dim3 blockDim(N, 1);
  dim3 gridDim(1);
  cudaLinearForward<<<gridDim, blockDim, sizeof(float) * N>>>(in_channels, out_channels, input, weight, bias, output, N);
}

void Linear::backward(float* input, float* src_error) {
  hipMemset(src_error, 0, sizeof(float) * in_channels);

  dim3 blockDim(out_channels, 1);
  dim3 gridDim(1);
  cudaLinearBackward<<<gridDim, blockDim>>>(in_channels, out_channels, input, weight, u_weight, u_bias, output, error, src_error);
}

void Linear::update(float rate) {
  dim3 blockDim(BLOCK_SIZE, 1);
  dim3 gridDim((in_channels * out_channels + BLOCK_SIZE - 1) / BLOCK_SIZE);
  cudaLinearUpdate<<<gridDim, blockDim>>>(in_channels, out_channels, weight, u_weight, bias, u_bias, error, rate);
}
